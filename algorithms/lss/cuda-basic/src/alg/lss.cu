#include "hip/hip_runtime.h"
/* Julian Gutierrez
 * Northeastern University
 * High Performance Computing
 * 
 * Level Set Segmentation for Image Processing 
 *  
 */
 
#include "lss.h"

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != hipSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
			exit(-1);
		}
	#endif
		return result;
}

using namespace std;

void modMaxIter (int value){
	max_iterations = value;
}

/*
 * Lss Step 1 from Pseudo Code
 */
__global__ void lssStep1(unsigned int* intensity, 
			 unsigned int* labels,
			 signed int* phi, 
			 int targetLabel, 
			 int lowerIntensityBound, 
			 int upperIntensityBound,
			 int* globalBlockIndicator) {

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int blockId = by*gridDim.x+bx;
				
	// Including border
	__shared__ signed char phiTile[TILE_SIZE][TILE_SIZE]; // output
	
	// Global Block Indicator
	__shared__ volatile signed int localGBI;
		
	// Read Input Data into Shared Memory
	/////////////////////////////////////////////////////////////////////////////////////

	int x = bx*TILE_SIZE+tx;
	int y = by*TILE_SIZE+ty;
	  
	int location = 	y*(gridDim.x*TILE_SIZE)+x;
		
	int intensityData = intensity[location];
	int     labelData = labels[location];
	
	localGBI = 0;	
	__syncthreads();
	
	// Algorithm 
	/////////////////////////////////////////////////////////////////////////////////////
	
	// Initialization
	if(intensityData >= lowerIntensityBound && 
	   intensityData <= upperIntensityBound) {
		if (labelData == targetLabel){
			phiTile[ty][tx] = 1;
		} else {
			phiTile[ty][tx] = -1;
			localGBI = 1;
		}
	} else {
		if (labelData == targetLabel){
			phiTile[ty][tx] = 2;
			localGBI = 1;
		} else {
			phiTile[ty][tx] = -2;
		}
	}
	
	__syncthreads();
	
	// Write back to main memory
	phi[location] = phiTile[ty][tx];
	
	if (tx == 0 && ty == 0){
		globalBlockIndicator[blockId]=localGBI;
	}

}

/*
 * Lss Step 2 from Pseudo Code
 */
 __global__ void lssStep2(signed int* phi, 
			 int* globalBlockIndicator,
			 int* globalFinishedVariable){

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	
	int blockId = by*gridDim.x+bx;
	
	// Including border
	__shared__ signed char    phiTile[TILE_SIZE+2][TILE_SIZE+2]; // input/output

	// Flags
	__shared__ volatile char BlockChange;
	__shared__ volatile char change;
	__shared__ volatile int redoBlock;
		
	// Read Global Block Indicator from global memory
	int localGBI = globalBlockIndicator[blockId];
	
	// Set Block Variables
	redoBlock = 0;
	
	__syncthreads();
	
	if (localGBI) {
		// Read Input Data into Shared Memory
		/////////////////////////////////////////////////////////////////////////////

		int x = bx*BLOCK_TILE_SIZE+tx;
		int y = by*BLOCK_TILE_SIZE+ty;
		  
		int location = 	y*(gridDim.x*BLOCK_TILE_SIZE)+x;
			
		int sharedX = tx+1;
		int sharedY = ty+1;
			
		phiTile[sharedY][sharedX] = phi[location];

		// Read Border Data into Shared Memory
		/////////////////////////////////////////////////////////////////////////////////////
		int posX;
		int posY;
		
		// Horizontal Border
		if (ty == 0){
			posX = sharedX;
			posY = 0;
			if (by == 0){
				phiTile[posY][posX] = -2;
			} else {
				phiTile[posY][posX] = phi[(y-1)*(gridDim.x*BLOCK_TILE_SIZE)+x];		
			}
		} else if (ty == BLOCK_TILE_SIZE-1){
			posX = sharedX;
			posY = BLOCK_TILE_SIZE+1;
			if (by == gridDim.y-1){
				phiTile[posY][posX] = -2;
			} else {
				phiTile[posY][posX] = phi[(y+1)*(gridDim.x*BLOCK_TILE_SIZE)+x];
			}
		}
		
		// Vertical Border
		if (tx == 0){
			posX = 0;
			posY = sharedY;
			if (bx == 0){
				phiTile[posY][posX] = -2;
			} else {
				phiTile[posY][posX] = phi[y*(gridDim.x*BLOCK_TILE_SIZE)+(x-1)];		
			}
		} else if (tx == BLOCK_TILE_SIZE-1){
			posX = BLOCK_TILE_SIZE+1;
			posY = sharedY;
			if (bx == gridDim.x-1){
				phiTile[posY][posX] = -2;
			} else {
				phiTile[posY][posX] = phi[y*(gridDim.x*BLOCK_TILE_SIZE)+(x+1)];
			}
		}
		
		BlockChange = 0; // Shared variable
		change      = 1; // Shared variable
		__syncthreads();
		
		// Algorithm 
		/////////////////////////////////////////////////////////////////////

		while (change){
			__syncthreads();
			change = 0;
			__syncthreads();
			
			if((phiTile[sharedY+1][sharedX]  == 1 ||
			    phiTile[sharedY-1][sharedX]  == 1 ||
			    phiTile[sharedY][sharedX+1]  == 1 ||
			    phiTile[sharedY][sharedX-1]  == 1 ) && 
			    phiTile[sharedY][sharedX]  == -1){
				phiTile[sharedY][sharedX] = 1;
				change = 1;
				BlockChange = 1;
			} else if ((phiTile[sharedY+1][sharedX]  == -2 ||
			    phiTile[sharedY-1][sharedX]  == -2 ||
			    phiTile[sharedY][sharedX+1]  == -2 ||
			    phiTile[sharedY][sharedX-1]  == -2 ) && 
			    phiTile[sharedY][sharedX]  == 2){
				phiTile[sharedY][sharedX] = -2;
				change = 1;
				BlockChange = 1;
			}
			__syncthreads();
		}
		
		if( phiTile[sharedY][sharedX]  == -1 || phiTile[sharedY][sharedX] == 2){
			redoBlock = 1;
		}
			
		__syncthreads();
		
		// Return value
		
		phi[location] = phiTile[sharedY][sharedX];
			
		if (tx == 0 && ty == 0) {
			if (BlockChange){
				*globalFinishedVariable = 1;
				__threadfence();
			}
			globalBlockIndicator[blockId] = redoBlock;
		}		
	}
}

/*
 * Lss Step 3 from Pseudo Code
 */
__global__ void lssStep3(signed int* phi,
			 signed int* phiOut) {

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;
				
	// Including border
	__shared__ signed int    phiTile[TILE_SIZE+2][TILE_SIZE+2]; // input
	__shared__ signed int phiOutTile[TILE_SIZE+2][TILE_SIZE+2]; // output

	// Read Input Data into Shared Memory
	/////////////////////////////////////////////////////////////////////////////////////

	int x = bx*BLOCK_TILE_SIZE+tx;
	int y = by*BLOCK_TILE_SIZE+ty;
	  
	int location = 	y*(gridDim.x*BLOCK_TILE_SIZE)+x;
		
	int sharedX = tx+1;
	int sharedY = ty+1;
		
	phiTile[sharedY][sharedX] = phi[location];

	// Read Border Data into Shared Memory
	/////////////////////////////////////////////////////////////////////////////////////
	int posX;
	int posY;
	
	// Horizontal Border
	if (ty == 0){
		posX = sharedX;
		posY = 0;
		if (by == 0){
			phiTile[posY][posX] = 0;
		} else {
			phiTile[posY][posX] = phi[(y-1)*(gridDim.x*BLOCK_TILE_SIZE)+x];		
		}
	} else if (ty == BLOCK_TILE_SIZE-1){
		posX = sharedX;
		posY = BLOCK_TILE_SIZE+1;
		if (by == gridDim.y-1){
			phiTile[posY][posX] = 0;
		} else {
			phiTile[posY][posX] = phi[(y+1)*(gridDim.x*BLOCK_TILE_SIZE)+x];
		}
	}
	
	// Vertical Border
	if (tx == 0){
		posX = 0;
		posY = sharedY;
		if (bx == 0){
			phiTile[posY][posX] = 0;
		} else {
			phiTile[posY][posX] = phi[y*(gridDim.x*BLOCK_TILE_SIZE)+(x-1)];		
		}
	} else if (tx == BLOCK_TILE_SIZE-1){
		posX = BLOCK_TILE_SIZE+1;
		posY = sharedY;
		if (bx == gridDim.x-1){
			phiTile[posY][posX] = 0;
		} else {
			phiTile[posY][posX] = phi[y*(gridDim.x*BLOCK_TILE_SIZE)+(x+1)];
		}
	}
		
	__syncthreads();
	
	// Algorithm 
	/////////////////////////////////////////////////////////////////////////////////////

	if(phiTile[sharedY][sharedX] > 0) {
		if(phiTile[sharedY+1][sharedX]  > 0 &&
		   phiTile[sharedY-1][sharedX]  > 0 &&
		   phiTile[sharedY][sharedX+1]  > 0 &&
		   phiTile[sharedY][sharedX-1]  > 0 ){
			phiOutTile[sharedY][sharedX] = 0xFD;
		} else 
			phiOutTile[sharedY][sharedX] = 0xFF;
	} else
		if(phiTile[sharedY+1][sharedX]  > 0 ||
		   phiTile[sharedY-1][sharedX]  > 0 ||
		   phiTile[sharedY][sharedX+1]  > 0 ||
		   phiTile[sharedY][sharedX-1]  > 0 ){
			phiOutTile[sharedY][sharedX] = 1;
		} else 
			phiOutTile[sharedY][sharedX] = 3;
	
	// Write back to main memory
	phiOut[location] = phiOutTile[sharedY][sharedX];
}

__global__ void evolveContour(unsigned int* intensity, 
			      unsigned int* labels,
			      signed int* phi,
			      signed int* phiOut, 
			      int gridXSize,
			      int gridYSize,
			      int* targetLabels, 
			      int* lowerIntensityBounds, 
			      int* upperIntensityBounds,
			      int max_iterations, 
			      int* globalBlockIndicator,
			      int* globalFinishedVariable,
			      int* totalIterations ) {
        int tid = threadIdx.x;

	// Setting up streams for 
	hipStream_t stream;
	hipStreamCreateWithFlags (&stream, hipStreamNonBlocking);
		
	// Total iterations
	totalIterations = &totalIterations[tid];
	
	// Size in ints
	int size = (gridXSize*BLOCK_TILE_SIZE)*(gridYSize*BLOCK_TILE_SIZE);
	
	// New phi pointer for each label.
	phi    = &phi[tid*size];
	phiOut = &phiOut[tid*size];

	globalBlockIndicator = &globalBlockIndicator[tid*gridXSize*gridYSize];

	// Global synchronization variable
	globalFinishedVariable = &globalFinishedVariable[tid];
	
	dim3 dimGrid(gridXSize, gridYSize);
        dim3 dimBlock(BLOCK_TILE_SIZE, BLOCK_TILE_SIZE);
	
	// Initialize phi array
	lssStep1<<<dimGrid, dimBlock, 0, stream>>>(intensity, 
					labels,  
					phi, 
					targetLabels[tid], 
					lowerIntensityBounds[tid], 
					upperIntensityBounds[tid],
					globalBlockIndicator);
	int iterations = 0;
	do {
		iterations++;
		lssStep2<<<dimGrid, dimBlock, 0, stream>>>(phi, 
					globalBlockIndicator,
					globalFinishedVariable );
		hipDeviceSynchronize();
	} while (atomicExch(globalFinishedVariable,0) && (iterations < max_iterations));
	
	lssStep3<<<dimGrid, dimBlock, 0, stream>>>(phi,
					phiOut);
	
	*totalIterations = iterations;
}

signed int *levelSetSegment(unsigned int *intensity, 
			    unsigned int *labels,
			    int height, 
			    int width,
			    int *targetLabels, 
			    int *lowerIntensityBounds,
			    int *upperIntensityBounds,
			    int numLabels){
	
	#if defined(DEBUG)
		printf("Printing input data\n");
		printf("Height: %d\n", height);
		printf("Width: %d\n", width);
		printf("Num Labels: %d\n", numLabels);
		
		for (int i = 0; i < numLabels; i++){
			printf("target label: %d\n", targetLabels[i]);
			printf("lower bound: %d\n", lowerIntensityBounds[i]);
			printf("upper bound: %d\n", upperIntensityBounds[i]);	
		}
	#endif
	
	int gridXSize = 1 + (( width - 1) / BLOCK_TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / BLOCK_TILE_SIZE);
	
	int XSize = gridXSize*BLOCK_TILE_SIZE;
	int YSize = gridYSize*BLOCK_TILE_SIZE;
	
	// Both are the same size (CPU/GPU).
	gpu.size = XSize*YSize;
	
	// Allocate arrays in GPU memory
	#if defined(VERBOSE)
		printf ("Allocating arrays in GPU memory.\n");
	#endif
	
	#if defined(CUDA_TIMING)
		float Ttime;
		TIMER_CREATE(Ttime);
		TIMER_START(Ttime);
	#endif
	
	checkCuda(hipMalloc((void**)&gpu.targetLabels           , numLabels*sizeof(int)));
        checkCuda(hipMalloc((void**)&gpu.lowerIntensityBounds   , numLabels*sizeof(int)));
        checkCuda(hipMalloc((void**)&gpu.upperIntensityBounds   , numLabels*sizeof(int)));
	checkCuda(hipMalloc((void**)&gpu.intensity              , gpu.size*sizeof(int)));
	checkCuda(hipMalloc((void**)&gpu.labels                 , gpu.size*sizeof(int)));
	checkCuda(hipMalloc((void**)&gpu.phi                    , numLabels*gpu.size*sizeof(int)));
	checkCuda(hipMalloc((void**)&gpu.phiOut                 , numLabels*gpu.size*sizeof(int)));
	checkCuda(hipMalloc((void**)&gpu.globalBlockIndicator   , numLabels*gridXSize*gridYSize*sizeof(int)));
	checkCuda(hipMalloc((void**)&gpu.globalFinishedVariable , numLabels*sizeof(int)));
	checkCuda(hipMalloc((void**)&gpu.totalIterations        , numLabels*sizeof(int)));
	
	// Allocate result array in CPU memory
	gpu.phiOnCpu = new signed int[gpu.size*numLabels];
	gpu.totalIterationsOnCpu = new int [numLabels];
	
        checkCuda(hipMemcpy(gpu.targetLabels, 
			targetLabels, 
			numLabels*sizeof(int), 
			hipMemcpyHostToDevice));

        checkCuda(hipMemcpy(gpu.lowerIntensityBounds, 
			lowerIntensityBounds, 
			numLabels*sizeof(int), 
			hipMemcpyHostToDevice));

        checkCuda(hipMemcpy(gpu.upperIntensityBounds, 
			upperIntensityBounds, 
			numLabels*sizeof(int), 
			hipMemcpyHostToDevice));
			
        checkCuda(hipMemcpy(gpu.intensity, 
			intensity, 
			gpu.size*sizeof(int), 
			hipMemcpyHostToDevice));
			
        checkCuda(hipMemcpy(gpu.labels, 
			labels, 
			gpu.size*sizeof(int), 
			hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());

	#if defined(CUDA_TIMING)
		float Ktime;
		TIMER_CREATE(Ktime);
		TIMER_START(Ktime);
	#endif
	
	#if defined(VERBOSE)
		printf("Running algorithm on GPU.\n");
	#endif
	
	// Launch kernel to begin image segmenation
	evolveContour<<<1, numLabels>>>(gpu.intensity, 
					gpu.labels,
					gpu.phi,
					gpu.phiOut, 
					gridXSize,
					gridYSize, 
					gpu.targetLabels, 
					gpu.lowerIntensityBounds, 
					gpu.upperIntensityBounds,
					max_iterations,
					gpu.globalBlockIndicator,
					gpu.globalFinishedVariable,
					gpu.totalIterations);
	
	checkCuda(hipDeviceSynchronize());

	#if defined(CUDA_TIMING)
		TIMER_END(Ktime);
		printf("Kernel Execution Time: %f ms\n", Ktime);
	#endif
	
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(gpu.phiOnCpu, 
			gpu.phiOut, 
			numLabels*gpu.size*sizeof(int), 
			hipMemcpyDeviceToHost));
	
	checkCuda(hipMemcpy(gpu.totalIterationsOnCpu, 
			gpu.totalIterations, 
			numLabels*sizeof(int), 
			hipMemcpyDeviceToHost));
			
	// Free resources and end the program
	checkCuda(hipFree(gpu.intensity));
	checkCuda(hipFree(gpu.labels));
	checkCuda(hipFree(gpu.phi));
	checkCuda(hipFree(gpu.phiOut));
	checkCuda(hipFree(gpu.targetLabels));
	checkCuda(hipFree(gpu.lowerIntensityBounds));
	checkCuda(hipFree(gpu.upperIntensityBounds));
	checkCuda(hipFree(gpu.globalBlockIndicator));
	checkCuda(hipFree(gpu.globalFinishedVariable));
	
	#if defined(CUDA_TIMING)
		TIMER_END(Ttime);
		printf("Total GPU Execution Time: %f ms\n", Ttime);
	#endif
	
	#if defined(VERBOSE)
		for (int i = 0; i < numLabels; i++){
			printf("target label: %d converged in %d iterations.\n", 
					targetLabels[i],
					gpu.totalIterationsOnCpu[i]);	
		}
	#endif
	
	return(gpu.phiOnCpu);

}
